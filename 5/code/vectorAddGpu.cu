#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"


void fillVector(int * v, size_t n);
void addVector(int * a, int *b, int *c, size_t n);
void printVector(int * v, size_t n);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b);


int main()
{
	const int vectorSize = 1024;
	int a[vectorSize], b[vectorSize], c[vectorSize];
	
	fillVector(a, vectorSize);
	fillVector(b, vectorSize);
	
	addWithCuda(c, a, b, vectorSize);
	printVector(c, vectorSize);
	
	return EXIT_SUCCESS;
}

// Fills a vector with data
void fillVector(int * v, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		v[i] = i;
	}
}


// Prints a vector to the stdout.
void printVector(int * v, size_t n) {
	int i;
	printf("[-] Vector elements: ");
	for (i = 0; i < n; i++) {
		printf("%d, ", v[i]);
	}
	printf("\b\b  \n");
}


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}
	
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	
	hipEventRecord(start);
	addKernel <<<1, 1024>>>(dev_c, dev_a, dev_b);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}
	
	float elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("elapsed_time : %f", elapsed_time);
	
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	return cudaStatus;
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
